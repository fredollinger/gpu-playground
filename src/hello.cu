#include "hip/hip_runtime.h"
// Adapted from here: https://www.amazon.com/CUDA-Application-Design-Development-Farber/dp/0123884268
// Example 1.7

#include <iostream>

using namespace std;

#include <thrust/reduce.h>
#include  <thrust/sequence.h>
#include  <thrust/host_vector.h>
#include  <thrust/device_vector.h>

__global__ void fillKernel(int *a, int n) {
 int tid = blockIdx.x * blockDim.x + threadIdx.x;
 if (tid < n) a[tid] = tid;
}

int fill(int *d_a, int n) {
    int nThreadsPerBlock = 12;
    int nBlocks = n/nThreadsPerBlock + ((n%nThreadsPerBlock) ? 1 : 0);
    fillKernel <<< nBlocks, nThreadsPerBlock >>> (d_a, n);
}


int main() {}